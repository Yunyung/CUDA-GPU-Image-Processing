#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

unsigned char *pdata; // pointer to data content

void getInfo(int *width, int *height, int *dataOffset, int *pixLen) {

    FILE *f;

    if (NULL == (f = fopen("lena_color.bmp", "rb"))) {

        printf("Fail to open the file1");

        exit(EXIT_FAILURE);

    }

    fseek(f, 0x00A, SEEK_SET);

    fread(dataOffset, sizeof(char) * 4, 1, f);

    fseek(f, 0x012, SEEK_SET );

    fread(width, sizeof(char) * 4, 1, f);

    fseek(f, 0x016, SEEK_SET);

    fread(height, sizeof(char) * 4, 1, f);

    fseek(f, 0x01C, SEEK_SET);

    fread(pixLen, sizeof(char)* 2, 1, f);
    *pixLen /= 8; //bit to byte
    printf("width = %d, height = %d, dataOffset = %d, pixLen = %d\n", *width, *height, *dataOffset, *pixLen);
    fclose(f);

}

void getData(int width, int height, int dataOffset, int pixLen) {
    FILE *f;

    if (NULL == (f = fopen("lena_color.bmp", "rb"))) {
    	
        printf("Fail to open the file2");

        exit(EXIT_FAILURE);

    }

    fseek(f, dataOffset, SEEK_SET);

    int size = fread(pdata, sizeof(unsigned char), width * height * pixLen, f);
    printf("Data size = %d byte \n", size);

    fclose(f);
}

void copy() {

    FILE *r, *w;

    unsigned char buf[1024];

    if (NULL == (r = fopen("lena_color.bmp", "rb"))) {

        printf("Fail to open the file3");

        exit(EXIT_FAILURE);

    }

    if (NULL == (w = fopen("result.bmp", "wb"))) {

        printf("Fail to open the file4");

        exit(EXIT_FAILURE);

    }

    

    while((fread(buf,sizeof(char),1024,r))>0)
        fwrite(buf,sizeof(char),1024,w);

    fclose(r);

    fclose(w);

}

void writeDataToImg(int width, int height, int dataOffset, int pixLen) {

    FILE *f;

    if (NULL == (f = fopen("result.bmp", "r+b"))) {

        printf("Fail to open the file5");

        exit(EXIT_FAILURE);

    }

    fseek(f, dataOffset, SEEK_SET);

    fwrite(pdata, sizeof(unsigned char), width * height * pixLen, f);

    fclose(f);

}

__global__ void processData(unsigned char *Da, int* filter)
{
    int tx = threadIdx.x;           // thread的x軸id
    int bx = blockIdx.x;            // block的x軸id
    int bn = blockDim.x;  
    int gid = bx * bn + tx;
    __shared__ int sfilter[3][3];
    __shared__ int sR[3][512];      // 每個block存上中下三行
    __shared__ int sG[3][512];
    __shared__ int sB[3][512];
    __shared__ int sRsum[512];      // 每個block 最後512個sum
    __shared__ int sGsum[512];
    __shared__ int sBsum[512];

    if (tx < 9)                     // 每個block 存filter 到 share memory
    {
        sfilter[tx / 3][tx % 3] = filter[tx];
    }
    __syncthreads();

    if (bx == 0 || bx == 511 || tx == 0 || tx == 511)
    {
        // 邊界處理 --> 直接給原本值不動
        sRsum[tx] = Da[gid * 3];
        sGsum[tx] = Da[gid * 3 + 1];
        sBsum[tx] = Da[gid * 3 + 2];
    }

    // 邊界處理(第1個block跟最後一個block不做)
    if (bx != 0 && bx != 511)
    {
    	// R, G, B個別將該Row(Block)運算會用到的上中下三行存入Share Memory
    	sR[0][tx] = Da[gid * 3 - 512 * 3];
		sR[1][tx] = Da[gid * 3];
		sR[2][tx] = Da[gid * 3 + 512 * 3];

		sG[0][tx] = Da[gid * 3 - 512 * 3 + 1];
		sG[1][tx] = Da[gid * 3 + 1];
		sG[2][tx] = Da[gid * 3 + 512 * 3 + 1];

		sB[0][tx] = Da[gid * 3 - 512 * 3 + 2];
		sB[1][tx] = Da[gid * 3 + 2];
		sB[2][tx] = Da[gid * 3 + 512 * 3 + 2];
		__syncthreads();

		// 邊界處理(每個block的的第一個值和最後一個值不做)
		if (tx != 0 && tx != 511)
		{
			// R
			sRsum[tx] = sR[0][tx - 1] * sfilter[0][0];
			sRsum[tx] += sR[0][tx] * sfilter[0][1];
			sRsum[tx] += sR[0][tx + 1] * sfilter[0][2];

			sRsum[tx] += sR[1][tx - 1] * sfilter[1][0];
			sRsum[tx] += sR[1][tx] * sfilter[1][1];
			sRsum[tx] += sR[1][tx + 1] * sfilter[1][2];

			sRsum[tx] += sR[2][tx - 1] * sfilter[2][0];
			sRsum[tx] += sR[2][tx] * sfilter[2][1];
			sRsum[tx] += sR[2][tx + 1] * sfilter[2][2];

			// G
			sGsum[tx] = sG[0][tx - 1] * sfilter[0][0];
			sGsum[tx] += sG[0][tx] * sfilter[0][1];
			sGsum[tx] += sG[0][tx + 1] * sfilter[0][2];

			sGsum[tx] += sG[1][tx - 1] * sfilter[1][0];
			sGsum[tx] += sG[1][tx] * sfilter[1][1];
			sGsum[tx] += sG[1][tx + 1] * sfilter[1][2];

			sGsum[tx] += sG[2][tx - 1] * sfilter[2][0];
			sGsum[tx] += sG[2][tx] * sfilter[2][1];
			sGsum[tx] += sG[2][tx + 1] * sfilter[2][2];

			// B
			sBsum[tx] = sB[0][tx - 1] * sfilter[0][0];
			sBsum[tx] += sB[0][tx] * sfilter[0][1];
			sBsum[tx] += sB[0][tx + 1] * sfilter[0][2];

			sBsum[tx] += sB[1][tx - 1] * sfilter[1][0];
			sBsum[tx] += sB[1][tx] * sfilter[1][1];
			sBsum[tx] += sB[1][tx + 1] * sfilter[1][2];

			sBsum[tx] += sB[2][tx - 1] * sfilter[2][0];
			sBsum[tx] += sB[2][tx] * sfilter[2][1];
			sBsum[tx] += sB[2][tx + 1] * sfilter[2][2];


			sRsum[tx] /= filter[9];
			sGsum[tx] /= filter[9];
			sBsum[tx] /= filter[9];
			// 大於255 或 小於0處理
			if (sRsum[tx] > 255)
				sRsum[tx] = 255;
			else if (sRsum[tx] < 0)
				sRsum[tx] = 0;

			if (sGsum[tx] > 255)
				sGsum[tx] = 255;
			else if (sGsum[tx] < 0)
				sGsum[tx] = 0;

			if (sBsum[tx] > 255)
				sBsum[tx] = 255;
			else if (sBsum[tx] < 0)
				sBsum[tx] = 0;
		}
    }

    __syncthreads();
    
	// 將R, G, B三個陣列值合併寫回一維陣列，以利輸出到檔案
	Da[gid * 3] = sRsum[tx];
	Da[gid * 3 + 1] = sGsum[tx];
	Da[gid * 3 + 2] = sBsum[tx];
}

void ImgDataProcess(int width, int height, int pixLen){

    int DataSize = width * height * pixLen; // 512 * 512 * 3

    /* GPU config */
    unsigned char *Da;
    int f[10];
    int choose;
    // user choose
    printf("請選擇您要的圖片轉換:\n");
    printf("1.模糊化\n");
    printf("2.銳利化\n");
    printf("選擇:");
    scanf("%d", &choose);
   	if (choose == 1)
   	{
   		for (int i = 0;i < 9;i++)
   			f[i] = 1;
   		f[9] = 9; // 模糊化 存最後要除的值
   	}
   	else if (choose == 2)
   	{
   		f[0] = 0; f[1] = -1; f[2] = 0;
   		f[3] = -1; f[4] = 5; f[5] = -1;
   		f[6] = 0; f[7] = -1; f[8] = 0;

   		f[9] = 1; // 銳利化signal
   	}
   	else
   	{
   		printf("沒這選項88");
   		exit(1);
   	}

    int *filter;
    hipMalloc((void**)&Da, DataSize);          //  create memory for save cpu data in gpu memory 
    hipMalloc((void**)&filter, 10 * sizeof(int));
    hipMemcpy(Da, pdata, DataSize, hipMemcpyHostToDevice);
    hipMemcpy(filter, f, 10 * sizeof(int), hipMemcpyHostToDevice);

    //  #########計算時間 宣告
    hipEvent_t start,stop;      //宣告起始時間和結束時間
    hipEventCreate(&start);     //分配開始時間的紀錄空間
    hipEventCreate(&stop);      //分配結束時間的紀錄空間

     /** 開始計時 **/
    hipEventRecord(start, 0);       //將起始時間歸零並開始計算
    //-------------------
    
    // 處理資料
    dim3 block(512, 1, 1);
    dim3 grid(512, 1, 1);
    processData <<< grid, block >>> (Da, filter);
    hipDeviceSynchronize();
    //-------------------
    
    hipEventRecord(stop, 0);        //將結束時間歸零並開始計算
    /** 結束計時 **/

    /*time slapsed*/
    hipEventSynchronize(stop);
    float elaspedTime;
    hipEventElapsedTime(&elaspedTime, start, stop);
    printf("Exe time: %f\n", elaspedTime); //print time
    hipEventDestroy(start);
    hipEventDestroy(stop);
    /* #########timing end*/

    // 將資料寫回pdata
    hipMemcpy(pdata, Da, DataSize, hipMemcpyDeviceToHost);



    hipFree(Da);hipFree(filter);
}

// CPU 
void transfer(int width, int height, int pixLen) {
    int i, j;
    int R[512][512];
    int G[512][512];
    int B[512][512];
    for(i = 0;i < height;i++) {
        for (j = 0;j < width;j++)
        {
            R[i][j] = pdata[(i * width * 3) + (j * 3)];
            G[i][j] = pdata[(i * width * 3) + (j * 3 + 1)];
            B[i][j] = pdata[(i * width * 3) + (j * 3 + 2)];
        }
    }
    int Rsum;
    int Gsum;
    int Bsum;
    for (i = 0;i < height;i++)  
    {
        for (j = 0;j < width;j++)
        {
            Rsum = 0;
            Gsum = 0;
            Bsum = 0;
            if (i == 0 || j == 0 || i == height - 1 || j == width - 1) // 邊緣不處理
            {
                // pdata[(i * width * 3) + (j * 3)] = R[i][j];
                // pdata[(i * width * 3) + (j * 3 + 1)] = G[i][j];
                // pdata[(i * width * 3) + (j * 3 + 2)] = B[i][j];
                continue;
            }
            Rsum += R[i - 1][j - 1] + R[i - 1][j] + R[i - 1][j + 1];
            Rsum += R[i][j - 1] + R[i][j] + R[i][j + 1];
            Rsum += R[i + 1][j - 1] + R[i + 1][j] + R[i + 1][j + 1];

            Gsum += G[i - 1][j - 1] + G[i - 1][j] + G[i - 1][j + 1];
            Gsum += G[i][j - 1] + G[i][j] + G[i][j + 1];
            Gsum += G[i + 1][j - 1] + G[i + 1][j] + G[i + 1][j + 1];


            Bsum += B[i - 1][j - 1] + B[i - 1][j] + B[i - 1][j + 1];
            Bsum += B[i][j - 1] + B[i][j] + B[i][j + 1];
            Bsum += B[i + 1][j - 1] + B[i + 1][j] + B[i + 1][j + 1];
            Rsum /= 9;
            Gsum /= 9;
            Bsum /= 9;
            if (Rsum > 255)
                Rsum = 255;
            else if (Rsum < 0)
                Rsum = 0;
            if (Gsum > 255)
                Gsum = 255;
            else if (Gsum < 0)
                Gsum = 0;
            if (Bsum > 255)
                Bsum = 255;
            else if (Bsum < 0)
                Bsum = 0;
        }
    }
}

int main() {

    int height, width;
    int dataOffset, pixLen;

    getInfo(&width, &height, &dataOffset, &pixLen);

    pdata = (unsigned char *)malloc(sizeof(unsigned char) * height * width * pixLen);

    getData(width, height, dataOffset, pixLen);

	// cpu 版本
	// transfer(width, height, pixLen);

    // 改變原始資料內容(pdata改變)
    ImgDataProcess(width, height, pixLen);

    
    copy(); //copy an backup of "lena.bmp"
    writeDataToImg(width, height, dataOffset, pixLen); // 將資料寫入新圖

    free(pdata);
}